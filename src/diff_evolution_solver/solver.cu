#include "hip/hip_runtime.h"
#include "diff_evolution_solver/solver.cuh"
#include "diff_evolution_solver/decoder.cuh"
#include "diff_evolution_solver/debug.cuh"
#include "diff_evolution_solver/evolve.cuh"
#include "diff_evolution_solver/evaluate.cuh"
// #include "cart_pole/cart_pole_utils.cuh"
// #include "cart_pole/model.cuh"
// #include "cart_pole/evaluate.cuh"
#include "footstep/footstep_utils.cuh"
#include "footstep/model.cuh"
#include "utils/utils_fun.cuh"
#include <math.h>

namespace cudaprocess{

void CudaDiffEvolveSolver::MallocSetup(){
    CHECK_CUDA(hipSetDevice(gpu_device_));

    // GPU Device
    // CHECK_CUDA(hipMalloc(&decoder_, sizeof(CudaProblemDecoder)));
    CHECK_CUDA(hipMalloc(&evolve_data_, sizeof(CudaEvolveData)));
    CHECK_CUDA(hipMalloc(&new_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE>)));
    CHECK_CUDA(hipMalloc(&old_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE * 3>)));
    // CHECK_CUDA(hipMalloc(&new_cluster_vec_, sizeof(CudaVector<CudaParamIndividual, CUDA_SOLVER_POP_SIZE>)));
    // CHECK_CUDA(hipMalloc(&problem_, sizeof(Problem)));
    CHECK_CUDA(hipMalloc(&evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&last_fitness, sizeof(float)));
    CHECK_CUDA(hipMalloc(&terminate_flag, sizeof(int)));
    CHECK_CUDA(hipMalloc(&result, sizeof(CudaParamIndividual)));

    // objective, constraint, constraint_score, lambda, parameter matrix
    // CHECK_CUDA(hipMalloc(&constraint_matrix, row_constraint * col_constraint * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&objective_matrix, row_obj * col_obj * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&constraint_score, CUDA_SOLVER_POP_SIZE * row_constraint * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&lambda_matrix, row_lambda * col_lambda * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&param_matrix, (dims_ + 1) * CUDA_SOLVER_POP_SIZE * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&objective_Q_matrix, row_obj_Q * col_obj_Q * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&quad_matrix, CUDA_SOLVER_POP_SIZE * CUDA_SOLVER_POP_SIZE * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&quad_transform, row_obj_Q * CUDA_SOLVER_POP_SIZE * sizeof(float)));
    // CHECK_CUDA(hipMalloc(&quadratic_score, 1 * CUDA_SOLVER_POP_SIZE * sizeof(float)));
    

    // CPU Host
    // CHECK_CUDA(hipHostAlloc(&h_terminate_flag, sizeof(int), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&host_result, sizeof(CudaParamIndividual), hipHostMallocDefault));
    CHECK_CUDA(hipHostAlloc(&host_evolve_data_, sizeof(CudaEvolveData), hipHostMallocDefault));

    // CHECK_CUDA(hipHostAlloc(&host_evolve_data_, sizeof(CudaEvolveData), hipHostMallocDefault));

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG || DEBUG_PRINT_WARM_START_FLAG){
        CHECK_CUDA(hipHostAlloc(&host_new_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE>), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&host_old_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3>), hipHostMallocDefault));
    }
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
        // objective, constraint, constraint_score, lambda, parameter, score matrix
        // CHECK_CUDA(hipHostAlloc(&h_constraint_matrix, row_constraint * col_constraint * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&h_objective_matrix, row_obj * col_obj * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&host_constraint_score, CUDA_SOLVER_POP_SIZE * row_constraint * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&h_lambda_matrix, row_lambda * col_lambda * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&host_param_matrix, (dims_ + 1) * CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&h_objective_Q_matrix, row_obj_Q * col_obj_Q * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&host_evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));

        // CHECK_CUDA(hipHostAlloc(&host_quad_matrix, CUDA_SOLVER_POP_SIZE * CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&h_quad_transform, row_obj_Q * CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));
        // CHECK_CUDA(hipHostAlloc(&h_quadratic_score, 1 * CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault));
    }

    // !--------------- Footstep ---------------!

    CHECK_CUDA(hipMalloc(&footstep::d_E, footstep::row_E * footstep::col_E * sizeof(float)));
    CHECK_CUDA(hipMalloc(&footstep::d_F, footstep::row_F * footstep::col_F * sizeof(float)));

    CHECK_CUDA(hipMalloc(&footstep::bigE, footstep::row_bigE * footstep::col_bigE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&footstep::bigF, footstep::row_bigF * footstep::col_bigF * sizeof(float)));

    CHECK_CUDA(hipMalloc(&footstep::d_cluster_N_state, footstep::N * CUDA_SOLVER_POP_SIZE * footstep::state_dims * sizeof(float)));

    CHECK_CUDA(hipMalloc(&footstep::d_sol_state, footstep::state_dims * footstep::N * sizeof(float)));

    CHECK_CUDA(hipMalloc(&footstep::d_sol_score, 3 * sizeof(float)));  // all socre, objective score, constraint score

    CHECK_CUDA(hipHostAlloc(&footstep::h_sol_state, footstep::state_dims * footstep::N * sizeof(float), hipHostMallocDefault)); 

    CHECK_CUDA(hipHostAlloc(&footstep::h_sol_score, 3 * sizeof(float), hipHostMallocDefault)); 

    if (DEBUG_PRINT_FLAG || DEBUG_FOOTSTEP){
        // printf("Debug flags enabled, allocating host memory\n");
        printf("bigE size:%d\n", footstep::row_bigE * footstep::col_bigE);
        printf("bigF size:%d\n", footstep::row_bigF * footstep::col_bigF);
        
        CHECK_CUDA(hipHostAlloc(&footstep::h_bigE, footstep::row_bigE * footstep::col_bigE * sizeof(float), hipHostMallocDefault));  
        CHECK_CUDA(hipHostAlloc(&footstep::h_bigF, footstep::row_bigF * footstep::col_bigF * sizeof(float), hipHostMallocDefault));    

        // CHECK_CUDA(hipHostAlloc(&footstep::h_N_state, footstep::state_dims * footstep::N * sizeof(float), hipHostMallocDefault));
        CHECK_CUDA(hipHostAlloc(&host_evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float), hipHostMallocDefault)); 
    }
    cuda_utils_ = std::make_shared<CudaUtil>();

    cudamalloc_flag = true;
}

void CudaDiffEvolveSolver::InitDiffEvolveParam(float top, float d_top, float min_top, float diff, float d_diff, float min_diff, float scale_f, float prob_crossover){
    top_ = top;
    d_top_ = d_top;
    min_top_ = min_top;
    diff_ = diff;
    d_diff_ = d_diff;
    min_diff_ = min_diff;
    
    lshade_param_.scale_f = lshade_param_.scale_f1 = scale_f;
    lshade_param_.Cr = prob_crossover;

}

__global__ void InitCudaEvolveData(CudaEvolveData* evolve, CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3>* old_cluster_data, int pop_size){
    int idx = threadIdx.x;
    if (idx == 0) {
        evolve->problem_param.top_ratio = 0.;
        evolve->hist_lshade_param.scale_f = evolve->hist_lshade_param.scale_f1 = 0.6;
        evolve->hist_lshade_param.Cr = 0.9;
        // evolve->new_cluster_vec->len = pop_size;
        old_cluster_data->len = pop_size;
    }
    if (idx < pop_size){
        // initial the each parameter in old_cluster 
        for (int i = 0; i < CUDA_PARAM_MAX_SIZE; ++i){
            old_cluster_data->all_param[(idx + pop_size) * CUDA_PARAM_MAX_SIZE + i] = 0.f;
        }
        old_cluster_data->fitness[idx + pop_size] = CUDA_MAX_FLOAT;
        // old_cluster_data->objective_score[idx + pop_size] = CUDA_MAX_FLOAT;
        // old_cluster_data->constraint_score[idx + pop_size] = CUDA_MAX_FLOAT;
    }
}

void CudaDiffEvolveSolver::SetBoundary(){
    for(int i = 0; i < dims_; ++i){
        if (i % footstep::control_dims == 0){
            host_evolve_data_->lower_bound[i] = footstep::ux_lb;
            host_evolve_data_->upper_bound[i] = footstep::ux_ub;
        }
        else if(i % footstep::control_dims == 1){
            host_evolve_data_->lower_bound[i] = footstep::uy_lb;
            host_evolve_data_->upper_bound[i] = footstep::uy_ub;
        }
        else{
            host_evolve_data_->lower_bound[i] = footstep::utheta_lb;
            host_evolve_data_->upper_bound[i] = footstep::utheta_ub;
        }
        // printf("current %d low bound:%f\n", i, host_evolve_data_->lower_bound[i]);
    }
}

/**
 * CudaEvolveData* ----> CudaParamClusterData<T> *
 */
__global__ void UpdateClusterDataBasedEvolve(CudaEvolveData* evolve_data, CudaParamClusterData<CUDA_SOLVER_POP_SIZE>* new_cluster_data, int num_last_potential_sol){
    int idx = blockIdx.x;
    if (idx >= num_last_potential_sol)   return;
    ConvertCudaParam<CUDA_SOLVER_POP_SIZE>(new_cluster_data, &evolve_data->last_potential_sol.data[idx], idx, threadIdx.x);
}

/**
 * CudaParamClusterData ----> CudaParamIndividual * as output
 */
template <int T>
__global__ void UpdateVecParamBasedClusterData(CudaParamIndividual *output, CudaParamClusterData<T> *cluster_data){
    ConvertCudaParamRevert<T>(cluster_data, &output[blockIdx.x], blockIdx.x, threadIdx.x);
}

/**
 * CudaParamClusterData<T> * ---->  CudaEvolveData* 
 */
__global__ void UpdateEvolveWarmStartBasedClusterData(CudaEvolveData *evolve_data, CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3> *old_param){
    ConvertCudaParamRevert<CUDA_SOLVER_POP_SIZE*3>(old_param, &evolve_data->warm_start, 0, threadIdx.x);
}

__global__ void SaveNewParamAsOldParam(CudaParamClusterData<CUDA_SOLVER_POP_SIZE> *new_param, CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3> *old_param, int left_bound, int right_bound, int bias){
    int sol_id = blockIdx.x;
    if (sol_id < left_bound || sol_id >= right_bound)   return;
    ConvertCudaParamBetweenClusters<CUDA_SOLVER_POP_SIZE, CUDA_SOLVER_POP_SIZE*3>(new_param, old_param, sol_id, sol_id + bias, threadIdx.x);
}

__global__ void GenerativeRandSolNearBest(CudaEvolveData *evolve, CudaParamClusterData<CUDA_SOLVER_POP_SIZE> *new_param, float *uniform_data, int rand_idx, float delta_con, float delta_int, int size){
    int sol_id = blockIdx.x;
    int param_id = threadIdx.x;

    if (sol_id == 0 || sol_id >= size)  return;
    float upper_bound = evolve->upper_bound[param_id];
    float lower_bound = evolve->lower_bound[param_id];

    if (param_id < evolve->problem_param.con_var_dims){
        float rand_range = (upper_bound - lower_bound) * delta_con;

        // based on rand_range update the boundary
        upper_bound = min(upper_bound, new_param->all_param[param_id] + rand_range);
        lower_bound = max(lower_bound, new_param->all_param[param_id] - rand_range);
        
        // based on new boundary near parameter to generate the new parameter
        new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
    }
    else if(param_id < evolve->problem_param.int_var_dims){
        float rand_range = (upper_bound - lower_bound) * delta_int;

        // based on rand_range update the boundary
        upper_bound = min(upper_bound, new_param->all_param[param_id] + rand_range);
        lower_bound = max(lower_bound, new_param->all_param[param_id] - rand_range);
        
        // based on new boundary near parameter to generate the new parameter
        new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
    }
}

// __global__ void GenerativeHeuristicsParam(CudaEvolveData *evolve, CudaParamClusterData<CUDA_SOLVER_POP_SIZE> *new_param, float *uniform_data, int rand_idx, float delta_con, float delta_int, int heuristics_size){
//     __shared__ float2 expected_force;
//     int sol_id = blockIdx.x;
//     int param_id = threadIdx.x;

//     if(sol_id >= heuristics_size)  return;

//     if (threadIdx.x == 0 && blockIdx.x == 0){
//         float pos = cart_pole::current_state.x, speed = cart_pole::current_state.y, theta = cart_pole::current_state.z, dtheta = cart_pole::current_state.w;
//         float right_wall_pos = cart_pole::current_wall_pos.x, left_wall_pos = cart_pole::current_wall_pos.y;

//         float pole_pos1 = -(cart_pole::ll * __sinf(theta) - pos), pole_pos2 = (cart_pole::ll * __sinf(theta) - pos);
//         float lam1 = 0.0f, lam2 = 0.0f;
//         if(pole_pos1 >= cart_pole::d_right){
//             lam1 = cart_pole::k1 * (right_wall_pos - cart_pole::d_right);
//             // printf("contact right wall, force:%f\n",lam1);
//         }
//         else if(pole_pos2 <= -cart_pole::d_left){
//             lam2 = cart_pole::k2 * (left_wall_pos - cart_pole::d_left);
//             printf("contact left wall, force:%f\n",lam2);
//         }
//         expected_force.x = -(-dtheta*dtheta*cart_pole::ll*cart_pole::mp*__sinf(theta) + cart_pole::g*cart_pole::mp*__sinf(2*theta)/2 + lam1*__cosf(theta)*__cosf(theta) - lam1 - lam2*__cosf(theta)*__cosf(theta) + lam2);
//         expected_force.y = -(-dtheta*dtheta*cart_pole::ll*cart_pole::mp*cart_pole::mp*__sinf(2*theta)/2 + cart_pole::g*cart_pole::mc*cart_pole::mp*__sinf(theta) + cart_pole::g*cart_pole::mp*cart_pole::mp*__sinf(theta) + lam1*cart_pole::mc*__cosf(theta) - lam2*cart_pole::mc*__cosf(theta))/(cart_pole::mp *__cosf(theta));
//         // printf("expected force1:%f, expected force2:%f\n",expected_force.x, expected_force.y);
//     }

//     float upper_bound = evolve->upper_bound[param_id];
//     float lower_bound = evolve->lower_bound[param_id];
//     int half_heuristics_size = heuristics_size >> 1;
//     // float expect_u = (sol_id >= half_heuristics_size)? expected_force.x : expected_force.y;
//     float expect_u = (sol_id >= half_heuristics_size)? expected_force.x : expected_force.y;

//     if (param_id < evolve->problem_param.con_var_dims){
//         float rand_range = (upper_bound - lower_bound) * delta_con;

//         // based on rand_range update the boundary
//         upper_bound = min(upper_bound, expect_u + rand_range);
//         lower_bound = max(lower_bound, expect_u - rand_range);
        
//         // based on new boundary near parameter to generate the new parameter
//         new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
//     }
//     else if(param_id < evolve->problem_param.int_var_dims){
//         float rand_range = (upper_bound - lower_bound) * delta_int;

//         // based on rand_range update the boundary
//         upper_bound = min(upper_bound, expect_u + rand_range);
//         lower_bound = max(lower_bound, expect_u - rand_range);
        
//         // based on new boundary near parameter to generate the new parameter
//         new_param->all_param[sol_id * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 100 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
//     }
// }

// __global__ void GenerativeRandomParamFromLastSol(CudaEvolveData *evolve, CudaParamClusterData<CUDA_SOLVER_POP_SIZE> *new_param, float *uniform_data, int rand_idx, float delta_con, float delta_int, int size, int bias){
//     CudaParamIndividual last_sol = evolve->warm_start;

//     int sol_id = blockIdx.x;
//     int param_id = threadIdx.x;

//     if(sol_id >= size)  return;

//     float upper_bound = evolve->upper_bound[param_id];
//     float lower_bound = evolve->lower_bound[param_id];

//     if (param_id < evolve->problem_param.con_var_dims){
//         float rand_range = (upper_bound - lower_bound) * delta_con;

//         // based on rand_range update the boundary
//         upper_bound = min(upper_bound, last_sol.param[param_id] + rand_range);
//         lower_bound = max(lower_bound, last_sol.param[param_id] - rand_range);
        
//         // based on new boundary near parameter to generate the new parameter
//         new_param->all_param[(sol_id + bias) * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 110 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
//     }
//     else if(param_id < evolve->problem_param.int_var_dims){
//         float rand_range = (upper_bound - lower_bound) * delta_int;

//         // based on rand_range update the boundary
//         upper_bound = min(upper_bound, last_sol.param[param_id] + rand_range);
//         lower_bound = max(lower_bound, last_sol.param[param_id] - rand_range);
        
//         // based on new boundary near parameter to generate the new parameter
//         new_param->all_param[(sol_id + bias) * CUDA_PARAM_MAX_SIZE + param_id] = lower_bound + uniform_data[CUDA_SOLVER_POP_SIZE * 110 * CUDA_PARAM_MAX_SIZE + sol_id * CUDA_SOLVER_POP_SIZE + rand_idx + param_id] * (upper_bound - lower_bound);
//     }
// }

__global__ void LoadWarmStartResultForSolver(CudaEvolveData *evolve, CudaParamClusterData<CUDA_SOLVER_POP_SIZE> *new_param){
    ConvertCudaParam<CUDA_SOLVER_POP_SIZE>(new_param, &evolve->warm_start, blockIdx.x, threadIdx.x);
}

void CudaDiffEvolveSolver::WarmStart(){
    InitParameter<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, CUDA_SOLVER_POP_SIZE, new_cluster_data_, old_cluster_data_, random_center_->uniform_data_);
    printf("warm start\n");
    Evaluation(CUDA_SOLVER_POP_SIZE, 0);

//     SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);
//     // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
//     int half_pop_size = CUDA_SOLVER_POP_SIZE >> 1;
//     int quad_pop_size = CUDA_SOLVER_POP_SIZE >> 2;
//     // setting half of population based on expected force and add some noise
//     GenerativeHeuristicsParam<<<quad_pop_size, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 32, 0.01, 0.01, quad_pop_size);

    

//     if(last_sol_fitness < 100){
//         // GenerativeRandSolNearBest<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 16, 0.1, 0.1, CUDA_SOLVER_POP_SIZE);
//         GenerativeRandomParamFromLastSol<<<half_pop_size, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 48, 0.01, 0.01, half_pop_size, quad_pop_size);
//         // SaveNewParamAsOldParam<<<quad_pop_size, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, old_cluster_data_, half_pop_size, half_pop_size+quad_pop_size, half_pop_size);
//         // if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("USING LAST POTENTIAL SOL\n");
//         // int half_pop_size = CUDA_SOLVER_POP_SIZE >> 1;
//         // int quad_pop_size = CUDA_SOLVER_POP_SIZE >> 2;
//         // // one cluster generate one solution, each cluster works on one block. 
//         // // We need to generate quad_pop_size new solutions based on last potential solution, so init the new cluster in quad_pop_size grid.
//         // UpdateClusterDataBasedEvolve<<<quad_pop_size, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, last_potential_sol_.len);
//     }
//     // UpdateVecParamBasedClusterData<CUDA_SOLVER_POP_SIZE><<<CUDA_SOLVER_POP_SIZE, 16, 0, cuda_utils_->streams_[0]>>>(new_cluster_vec_->data, new_cluster_data_);

//     // // int cet = 10;
//     // // Update the output param based on warm start.
//     // // CHECK_CUDA(hipMemcpyAsync(output_sol, &new_cluster_vec_->data[cet], sizeof(CudaParamIndividual), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));

//     // // Evaluate random solutions or potential solutions in warmstart
//     Evaluation(CUDA_SOLVER_POP_SIZE, 0);

//     // // SortParamBasedBitonic<64><<<16, 64, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_->all_param, new_cluster_data_->fitness);

//     // Find the best solution among the random solutions or potential solutions in warmstart and put it in the first place
//     // ParaFindMax2<CUDA_SOLVER_POP_SIZE, CUDA_SOLVER_POP_SIZE><<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_);

//     // put heuristics individual in para_old[0, half_pop_size)
//     SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);

//     // // based on warm start result to generate random solution. Further improve the quality of the initial population
//     // GenerativeRandSolNearBest<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 16, 0.1, 0.1, CUDA_SOLVER_POP_SIZE);

//     // // convert the parameter from warm start to old parameter
//     // SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);

//     // // Based on all old parameter to update the warm start of evolve data
//     // // 将 old_cluster_data_<CUDA_SOLVER_POP_SIZE*3> 中索引为0的数据提取出来,填充到evolve data单个CudaParamIndividual结构中,记为warm start。索引为0的解是warm start过程中最优的
//     // UpdateEvolveWarmStartBasedClusterData<<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, old_cluster_data_);

//     if(DEBUG_PRINT_WARM_START_FLAG){
//         // CHECK_CUDA(hipMemcpyAsync(host_old_cluster_data_, old_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3>), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
//         CHECK_CUDA(hipMemcpyAsync(host_new_cluster_data_, new_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE>), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
//         CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
//         // PrintClusterData<CUDA_SOLVER_POP_SIZE*3>(host_old_cluster_data_);
//         PrintClusterData<CUDA_SOLVER_POP_SIZE>(host_new_cluster_data_);
//     }

//     // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
}

// (Abandoned) Use for loop to evaluate 
// template<int T>
// __global__ void MainEvaluation(CudaEvolveData *evolve, CudaParamClusterData<T> *cluster_data){
//     DynamicEvaluation2(evolve, cluster_data, evolve->lambda);
// }

auto max_dim3 = [](const dim3& a, const dim3& b) {
    return (a.x * a.y * a.z) > (b.x * b.y * b.z) ? a : b;
};

void CudaDiffEvolveSolver::Evaluation(int size, int epoch){
    printf("current epoch:%d\n",epoch);
    // CHECK_CUDA(hipDeviceSynchronize());
    const size_t gemm_shared_mem_size = std::max(footstep::Ex_GEMM().shared_memory_size, footstep::Fu_GEMM().shared_memory_size);

    dim3 dim = max_dim3(footstep::Ex_GEMM().block_dim, footstep::Fu_GEMM().block_dim);
    printf("cublasDx need share memory size:%zu max block dim:%u\n",gemm_shared_mem_size, dim.x*dim.y*dim.z); 
    
    // // 验证是否超过设备限制
    hipGetDevice(0);
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, 0));
    std::cout << "Device shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
    
    if (gemm_shared_mem_size > prop.sharedMemPerBlock) {
        std::cout << "Required shared memory (" << gemm_shared_mem_size 
                  << " bytes) exceeds device limit (" << prop.sharedMemPerBlock 
                  << " bytes)" << std::endl;

        // Increase max dynamic shared memory for the kernel if needed
        CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
            footstep::UpdateState<CUDA_SOLVER_POP_SIZE>), 
            hipFuncAttributeMaxDynamicSharedMemorySize,
            gemm_shared_mem_size
        ));
    }
    
    // hipGetDevice(0);  // 获取当前设备
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, 0);  // 获取设备属性

    // std::cout << "==================== CUDA Device Info ====================" << std::endl;
    // std::cout << "Device Name                        : " << prop.name << std::endl;
    // std::cout << "Max Shared Memory per Block        : " << prop.sharedMemPerBlock << " bytes" << std::endl;
    // std::cout << "Max Shared Memory per Multiprocessor: " << prop.sharedMemPerMultiprocessor << " bytes" << std::endl;
    // std::cout << "Max Threads per Block              : " << prop.maxThreadsPerBlock << std::endl;
    // std::cout << "Warp Size                           : " << prop.warpSize << std::endl;
    // std::cout << "=========================================================" << std::endl;
    CHECK_CUDA(hipMemset(footstep::d_cluster_N_state, 0, sizeof(footstep::d_cluster_N_state)));

    footstep::UpdateState<CUDA_SOLVER_POP_SIZE><<<size, dim, gemm_shared_mem_size, cuda_utils_->streams_[0]>>>(new_cluster_data_, footstep::bigE, footstep::bigF, footstep::d_cluster_N_state);

    footstep::EvaluateModel<CUDA_SOLVER_POP_SIZE><<<size, 32, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, footstep::d_cluster_N_state, evaluate_score_);

    if(DEBUG_PRINT_FLAG || DEBUG_FOOTSTEP){
        CHECK_CUDA(hipMemcpy(footstep::h_cluster_param, new_cluster_data_->all_param, CUDA_SOLVER_POP_SIZE * CUDA_PARAM_MAX_SIZE * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(footstep::h_cluster_N_state, footstep::d_cluster_N_state, footstep::N * CUDA_SOLVER_POP_SIZE * footstep::state_dims * sizeof(float), hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(host_evaluate_score_, evaluate_score_, CUDA_SOLVER_POP_SIZE * sizeof(float), hipMemcpyDeviceToHost));

        PrintMatrixByRow(footstep::h_cluster_param, CUDA_SOLVER_POP_SIZE , CUDA_PARAM_MAX_SIZE, "h_cluster_param");
        PrintMatrixByRow(footstep::h_cluster_N_state, CUDA_SOLVER_POP_SIZE , footstep::N * footstep::state_dims, "cluster_N_state");
        PrintMatrixByRow(host_evaluate_score_, CUDA_SOLVER_POP_SIZE , 1, "evaluation score");
    }
    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
    UpdateFitnessBasedMatrix<CUDA_SOLVER_POP_SIZE><<<1, size, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, evaluate_score_);
}

void CudaDiffEvolveSolver::Evolution(int epoch, CudaEvolveType search_type){
    DuplicateBestAndReorganize<<<CUDA_PARAM_MAX_SIZE, CUDA_SOLVER_POP_SIZE*3, 0, cuda_utils_->streams_[0]>>>(epoch, old_cluster_data_, 2);
    CudaEvolveProcess<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(epoch, old_cluster_data_, new_cluster_data_, random_center_->uniform_data_, random_center_->normal_data_, evolve_data_, CUDA_SOLVER_POP_SIZE, true);
    Evaluation(CUDA_SOLVER_POP_SIZE, epoch);

    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
    UpdateParameter<CUDA_SOLVER_POP_SIZE><<<CUDA_PARAM_MAX_SIZE, CUDA_SOLVER_POP_SIZE*2, 0, cuda_utils_->streams_[0]>>>(epoch, evolve_data_, new_cluster_data_, old_cluster_data_);

    // CHECK_CUDA(hipMemcpyAsync(h_terminate_flag, terminate_flag, sizeof(int), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
}

void CudaDiffEvolveSolver::InitSolver(int gpu_device){
    if(DEBUG_ENABLE_NVTX)   init_range = nvtxRangeStart("Init Different Evolution Solver");

    gpu_device_ = gpu_device;
    random_center_ =std::make_shared<CudaRandomManager>(gpu_device_);

    CHECK_CUDA(hipSetDevice(gpu_device_));
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("CUDA SET DEVICE\n");

    dims_ = footstep::control_dims * footstep::N;

    con_var_dims_ = dims_;
    int_var_dims_ = 0;

    // Initialize cuBLAS handle
    hipblasStatus_t status = hipblasCreate(&cublas_handle_);

    MallocSetup();

    footstep::ConstructEandF(cuda_utils_->streams_[0]);
    footstep::ConstructBigEAndF(footstep::bigE, footstep::bigF, cublas_handle_, cuda_utils_->streams_[0]);

    // CHECK_CUDA(hipMemcpy(footstep::d_init_state, footstep::init_state, footstep::state_dims * sizeof(float), hipMemcpyHostToDevice));

    // CHECK_CUDA(hipMemset(footstep::d_cluster_N_state, 0, sizeof(footstep::d_cluster_N_state)));

    if (DEBUG_PRINT_FLAG || DEBUG_FOOTSTEP){
        printf("Debug flags enabled, copying memory\n");
        CHECK_CUDA(hipMemcpyAsync(footstep::h_bigE, footstep::bigE, footstep::row_bigE * footstep::col_bigE * sizeof(float), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
        CHECK_CUDA(hipMemcpyAsync(footstep::h_bigF, footstep::bigF, footstep::row_bigF * footstep::col_bigF * sizeof(float), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
        CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));

        PrintMatrixByRow(footstep::h_bigE, footstep::row_bigE, footstep::col_bigE, "bigE:");
        PrintMatrixByRow(footstep::h_bigF, footstep::row_bigF, footstep::col_bigF, "bigF:");
    }

    InitDiffEvolveParam();
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("INIT PARAM FOR DE\n");
    
    // *h_terminate_flag = 0;
    // hipMemset(terminate_flag, 0, sizeof(int));
    // float init_last_f = CUDA_MAX_FLOAT;
    // CHECK_CUDA(hipMemcpy(last_fitness, &init_last_f, sizeof(float), hipMemcpyHostToDevice));

    if(DEBUG_ENABLE_NVTX)   setting_boundary_range = nvtxRangeStart("Init_Solver Setting Boundary");

    host_evolve_data_->problem_param.con_var_dims = con_var_dims_;
    host_evolve_data_->problem_param.dims = dims_;
    host_evolve_data_->problem_param.int_var_dims = int_var_dims_;

    host_evolve_data_->problem_param.max_round = 60;

    host_evolve_data_->problem_param.accuracy_rng = 0.5;

    SetBoundary();
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("START MEMORY ASYNC\n");

    CHECK_CUDA(hipMemcpyAsync(evolve_data_, host_evolve_data_, sizeof(CudaEvolveData), hipMemcpyHostToDevice, cuda_utils_->streams_[0]));

    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_INIT_SOLVER_FLAG) printf("MEMORY ASYNC SUBMIT\n");

    InitCudaEvolveData<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, old_cluster_data_, CUDA_SOLVER_POP_SIZE);

    // WarmStart();

    // CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));

    // size_t uniform_size = random_center_->uniform_size_;
    // std::vector<float> host_uniform(uniform_size);
    
    // // 拷贝所有数据到host
    // CHECK_CUDA(hipMemcpy(host_uniform.data(), random_center_->uniform_data_, 
    //           uniform_size * sizeof(float), hipMemcpyDeviceToHost));
    
    // printf("All uniform random numbers (total size: %zu):\n", uniform_size);
    // for(size_t i = 0; i < uniform_size; i++) {
    //     printf("uniform_data_[%zu] = %f\n", i, host_uniform[i]);
    //     // 验证数值是否合理
    //     if(host_uniform[i] < 0.0f || host_uniform[i] > 1.0f) {
    //         printf("WARNING: Invalid random number at %zu: %f\n", i, host_uniform[i]);
    //     }
    // }

    printf("FINISH INIT SOLVER\n");
}

// void CudaDiffEvolveSolver::UpdateCartPoleSystem(float sys_state[4], float wall_pos[2]){
//     float4 new_state = {sys_state[0], sys_state[1], sys_state[2], sys_state[3]};
//     float2 new_wall_pos = {wall_pos[0], wall_pos[1]};
    
//     CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(cart_pole::current_state), &new_state, sizeof(float4)));
//     CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(cart_pole::current_wall_pos), &new_wall_pos, sizeof(float2)));
//     hipDeviceSynchronize();
// }


template <int T=CUDA_SOLVER_POP_SIZE*3>
__global__ void GetSolFromOldParam(CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3> *old_param, CudaParamIndividual *solution){
    ConvertCudaParamRevert<CUDA_SOLVER_POP_SIZE*3>(old_param, solution, blockIdx.x, threadIdx.x);
}

CudaParamIndividual CudaDiffEvolveSolver::Solver(){
    // nvtx3::mark("Different Evolvution Solver!");
    if(DEBUG_ENABLE_NVTX)   solver_range = nvtxRangeStart("Different Evolvution Solver");

    // init_pop_size_ = CUDA_SOLVER_POP_SIZE;
    // pop_size_ = CUDA_SOLVER_POP_SIZE;

    InitCudaEvolveData<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, old_cluster_data_, CUDA_SOLVER_POP_SIZE);

    InitParameter<<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, CUDA_SOLVER_POP_SIZE, new_cluster_data_, old_cluster_data_, random_center_->uniform_data_);
    
    // if(enable_warmstart)    LoadWarmStartResultForSolver<<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_);

    // // based on warm start result to generate 
    // GenerativeRandSolNearBest<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, new_cluster_data_, random_center_->uniform_data_, 16, 0.001, 0.001, CUDA_SOLVER_POP_SIZE);
    
    Evaluation(CUDA_SOLVER_POP_SIZE, 0);

    ParaFindMax2<CUDA_SOLVER_POP_SIZE, CUDA_SOLVER_POP_SIZE><<<1, CUDA_SOLVER_POP_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_);

    SaveNewParamAsOldParam<<<CUDA_SOLVER_POP_SIZE, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(new_cluster_data_, old_cluster_data_, 0, CUDA_SOLVER_POP_SIZE, 0);
    
    // WarmStart();

    // float sol_obj_score = CUDA_MAX_FLOAT;
    // float sol_score = CUDA_MAX_FLOAT;
    // float *sol_state = nullptr;
    // bool satisify = false;
    for (int i = 0; i < host_evolve_data_->problem_param.max_round; ++i) {
        // printf("generation i:%d\n", i);
        Evolution(i, CudaEvolveType::GLOBAL);

        if(i == host_evolve_data_->problem_param.max_round - 1){
            printf("CHECK solution\n");
            const size_t gemm_shared_mem_size = std::max(footstep::Ex_GEMM().shared_memory_size, footstep::Fu_GEMM().shared_memory_size);

            dim3 dim = max_dim3(footstep::Ex_GEMM().block_dim, footstep::Fu_GEMM().block_dim);

            // // 验证是否超过设备限制
            hipGetDevice(0);
            hipDeviceProp_t prop;
            CHECK_CUDA(hipGetDeviceProperties(&prop, 0));
            std::cout << "Device shared memory per block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
            
            if (gemm_shared_mem_size > prop.sharedMemPerBlock) {
                std::cout << "Required shared memory (" << gemm_shared_mem_size 
                        << " bytes) exceeds device limit (" << prop.sharedMemPerBlock 
                        << " bytes)" << std::endl;

                // Increase max dynamic shared memory for the kernel if needed
                CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
                    footstep::UpdateState<CUDA_SOLVER_POP_SIZE>), 
                    hipFuncAttributeMaxDynamicSharedMemorySize,
                    gemm_shared_mem_size
                ));
            }
            CHECK_CUDA(hipMemset(footstep::d_sol_state, 0.0f, sizeof(footstep::d_sol_state)));
            footstep::UpdateState<CUDA_SOLVER_POP_SIZE*3><<<1, dim, gemm_shared_mem_size, cuda_utils_->streams_[0]>>>(old_cluster_data_, footstep::bigE, footstep::bigF, footstep::d_sol_state);
            footstep::EvaluateModel<CUDA_SOLVER_POP_SIZE*3><<<1, 32, 0, cuda_utils_->streams_[0]>>>(old_cluster_data_, footstep::d_sol_state, evaluate_score_, footstep::d_sol_score);

            CHECK_CUDA(hipMemcpy(footstep::h_sol_state, footstep::d_sol_state, footstep::N * footstep::state_dims * sizeof(float), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipMemcpy(footstep::h_sol_score, footstep::d_sol_score, 3 * sizeof(float), hipMemcpyDeviceToHost));
            CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
            printf("sol_score:%f, sol_obj_score:%f, sol_constraint_score:%f\n",footstep::h_sol_score[0], footstep::h_sol_score[1], footstep::h_sol_score[2]);
        }
    }
    
    if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG){
        CHECK_CUDA(hipMemcpyAsync(host_old_cluster_data_, old_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE*3>), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
        CHECK_CUDA(hipMemcpyAsync(host_new_cluster_data_, new_cluster_data_, sizeof(CudaParamClusterData<CUDA_SOLVER_POP_SIZE>), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
        CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
        PrintClusterData<CUDA_SOLVER_POP_SIZE*3>(host_old_cluster_data_);
        printf("new cluster data=============================================\n");
        PrintClusterData<CUDA_SOLVER_POP_SIZE>(host_new_cluster_data_);

        // CHECK_CUDA(hipMemcpyAsync(host_evolve_data_, evolve_data_, sizeof(CudaEvolveData), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
        // printf("CUDA_MAX_FLOAT %f\n", CUDA_MAX_FLOAT);
    }
    // Get the first individual from old param (after sorting, the first one is the best one)
    GetSolFromOldParam<CUDA_SOLVER_POP_SIZE*3><<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(old_cluster_data_, result);
    // // 将 old_cluster_data_<CUDA_SOLVER_POP_SIZE*3> 中索引为0的数据提取出来,填充到evolve data单个CudaParamIndividual结构中,记为warm start。索引为0的解是warm start过程中最优的
    // UpdateEvolveWarmStartBasedClusterData<<<1, CUDA_PARAM_MAX_SIZE, 0, cuda_utils_->streams_[0]>>>(evolve_data_, old_cluster_data_);
    CHECK_CUDA(hipMemcpyAsync(host_result, result, sizeof(CudaParamIndividual), hipMemcpyDeviceToHost, cuda_utils_->streams_[0]));
    CHECK_CUDA(hipStreamSynchronize(cuda_utils_->streams_[0]));
    // hipDeviceSynchronize();

    host_result->objective_score = footstep::h_sol_score[1];
    host_result->constraint_score = footstep::h_sol_score[2];
    host_result->N_states = footstep::h_sol_state;
    

    // for(int i = con_var_dims_; i < dims_; ++i){
    //     host_result->param[i] = floor(host_result->param[i]);
    // }

    // if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG)   printFinalResult(host_result->fitness, host_result->param, dims_);
    // printFinalResult(host_result->fitness, host_result->param, dims_);

    if(DEBUG_ENABLE_NVTX)   nvtxRangeEnd(solver_range);

    return *host_result;
}

CudaDiffEvolveSolver::~CudaDiffEvolveSolver(){
    if (cudamalloc_flag){
        // GPU device
        CHECK_CUDA(hipFree(evolve_data_));
        CHECK_CUDA(hipFree(new_cluster_data_));
        CHECK_CUDA(hipFree(old_cluster_data_));
        // CHECK_CUDA(hipFree(new_cluster_vec_));
        CHECK_CUDA(hipFree(constraint_matrix));
        CHECK_CUDA(hipFree(objective_matrix));
        CHECK_CUDA(hipFree(param_matrix));
        CHECK_CUDA(hipFree(evaluate_score_));
        CHECK_CUDA(hipFree(constraint_score));
        CHECK_CUDA(hipFree(quad_matrix));
        CHECK_CUDA(hipFree(quad_transform));
        CHECK_CUDA(hipFree(quadratic_score));
        CHECK_CUDA(hipFree(lambda_matrix));
        CHECK_CUDA(hipFree(result));

        // CPU host
        if (DEBUG_PRINT_FLAG || DEBUG_PRINT_SOLVER_FLAG){
            CHECK_CUDA(hipHostFree(host_new_cluster_data_));
            CHECK_CUDA(hipHostFree(host_old_cluster_data_));
        }

        if (DEBUG_PRINT_FLAG || DEBUG_PRINT_EVALUATE_FLAG){
            CHECK_CUDA(hipHostFree(host_evaluate_score_));
            CHECK_CUDA(hipHostFree(host_param_matrix));
            CHECK_CUDA(hipHostFree(host_constraint_score));
            CHECK_CUDA(hipHostFree(h_lambda_matrix));
            CHECK_CUDA(hipHostFree(h_constraint_matrix));
            CHECK_CUDA(hipHostFree(h_objective_matrix));
            CHECK_CUDA(hipHostFree(host_quad_matrix));
            CHECK_CUDA(hipHostFree(h_quad_transform));
            CHECK_CUDA(hipHostFree(h_quadratic_score));
        }
        
        CHECK_CUDA(hipHostFree(host_evolve_data_));
        CHECK_CUDA(hipHostFree(host_result));
        
    }
}
}