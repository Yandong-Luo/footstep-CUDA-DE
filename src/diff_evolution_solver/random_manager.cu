#include "diff_evolution_solver/random_manager.cuh"

namespace cudaprocess{
    CudaRandomManager::CudaRandomManager(int gpu_device) {
        CHECK_CUDA(hipSetDevice(gpu_device));
        CHECK_CUDA(hipStreamCreate(&stream));
        
        CHECK_CUDA(hipMalloc(&uniform_data_, sizeof(float) * uniform_size_));
        CHECK_CUDA(hipMalloc(&normal_data_, sizeof(float) * normal_size_));
        
        CURAND_CHECK(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW));
        CURAND_CHECK(hiprandSetStream(gen, stream));
        CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
        
        Generate();
    }

    CudaRandomManager::~CudaRandomManager() {
        hipStreamSynchronize(stream);
        CURAND_CHECK(hiprandDestroyGenerator(gen));
        CHECK_CUDA(hipFree(uniform_data_));
        CHECK_CUDA(hipFree(normal_data_));
        CHECK_CUDA(hipStreamDestroy(stream));
    }

    void CudaRandomManager::Generate() {
        CURAND_CHECK(hiprandGenerateUniform(gen, uniform_data_, uniform_size_));
        CURAND_CHECK(hiprandGenerateNormal(gen, normal_data_, normal_size_, 0.0f, 1.0f));
        hipStreamSynchronize(stream);
    }

    void CudaRandomManager::Regenerate(unsigned long long seed) {
        // 重置随机数生成器的种子
        CURAND_CHECK(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
        
        // 同步确保之前的操作完成
        hipStreamSynchronize(stream);
        
        // 重新生成随机数
        Generate();
    }

    void CudaRandomManager::Regenerate() {
        Regenerate(GenerateRandomSeed());
    }

    unsigned long long CudaRandomManager::GenerateRandomSeed() {
        // 方法1：使用时间戳和硬件随机数结合
        auto timestamp = std::chrono::high_resolution_clock::now().time_since_epoch().count();
        std::random_device rd;  // 硬件随机数生成器
        
        // 将时间戳和随机数组合生成种子
        unsigned long long seed = static_cast<unsigned long long>(timestamp) ^ 
                                (static_cast<unsigned long long>(rd()) << 32);
        return seed;
    }
}