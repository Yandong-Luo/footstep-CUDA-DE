#include "hip/hip_runtime.h"
#include "curve/bezier_curve.cuh"

namespace cudaprocess{

namespace bezier_curve{

float *d_all_curve_param = nullptr;
float *h_all_curve_param = nullptr;

// *************** Polar to Cartesian *****************
// __device__ __forceinline__ float2 PolarToCartesian(float radius, float theta) {
// 	return make_float2(radius * __cosf(theta), radius * __sinf(theta));
// }

// __device__ __forceinline__ float2 PolarVelocityToCartesian(float2 position, float2 velocity) {
// 	float radius = position.x, theta = position.y;
// 	float v_r = velocity.x, v_theta = velocity.y;
// 	float Vx = v_r * __cosf(theta) - radius * v_theta * __sinf(theta);
// 	float Vy = v_r * __sinf(theta) + radius * v_theta * __cosf(theta);
// 	return make_float2(Vx, Vy);
// }

// __device__ __forceinline__ void PolarAllState2Cartesian(float2 position, float2 velocity, float *state, int step_idx){
//     float radius = position.x, theta = position.y;
//     float v_r = velocity.x, v_theta = velocity.y;
// 	int idx = step_idx * 6;
// 	state[idx] = radius * __cosf(theta);
//     state[idx + 1] = radius * __sinf(theta);
//     state[idx + 2] = v_r * __cosf(theta) - radius * v_theta * __sinf(theta);
//     state[idx + 3] = v_r * __sinf(theta) + radius * v_theta * __cosf(theta);
//     state[idx + 4] = theta;
// 	state[idx + 5] = radius;
// }

// *************** Bezier *****************
__global__ void PrepareBinomialandFixedPoint(BezierCurve* curve) {
	float t_powers[BEZIER_SIZE], one_minus_t_powers[BEZIER_SIZE];	// record t^j, (1-t)^{n-j}

	float tmp_binomial_coeff[BEZIER_SIZE][BEZIER_SIZE + 1];
	
	tmp_binomial_coeff[0][0] = tmp_binomial_coeff[1][1] = 1;
	tmp_binomial_coeff[0][1] = 0;
	tmp_binomial_coeff[1][0] = 1;

	// Pascal’s Triangle to calculate combination Coefficients
	for (int i{2}; i < BEZIER_SIZE; ++i) {
		tmp_binomial_coeff[i][0] = tmp_binomial_coeff[i][i] = 1;
		for (int j{1}; j < i; ++j) {
			tmp_binomial_coeff[i][j] = tmp_binomial_coeff[i - 1][j - 1] + tmp_binomial_coeff[i - 1][j];		// C_j^i
		}
	}
	// save Binomial Coefficients to comb
	for (int i{0}; i < BEZIER_SIZE; ++i) {
		curve->binomial_coeff_[i] = tmp_binomial_coeff[BEZIER_SIZE - 1][i];
		// printf("combination coefficients for position:%d %f\n", i, curve->binomial_coeff_[i]);
	}

	const int num_steps = CURVE_NUM_STEPS - 1;
    const int n = BEZIER_SIZE - 1;  // 贝塞尔曲线的阶数
    
    // 预计算低一阶的二项式系数(用于导数计算)
    for (int i = 0; i < BEZIER_SIZE - 1; ++i) {
		curve->binomial_deriv_coeff_[i] = tmp_binomial_coeff[BEZIER_SIZE - 2][i];
    }

	for (int i = 0; i <= num_steps; ++i) {
		float t = i * (1.0f / num_steps);
		t_powers[0] = one_minus_t_powers[0] = 1;

		for (int j = 1; j < BEZIER_SIZE; ++j) {
			t_powers[j] = t_powers[j - 1] * t;
			one_minus_t_powers[j] = one_minus_t_powers[j - 1] * (1 - t);

			// printf("current step:%d j:%d t_powers:%f\n", i, j, t_powers[j]);
			// printf("current step:%d j:%d one_minus_t_powers:%f\n", i, j, one_minus_t_powers[j]);
		}

		for (int j = 0; j < BEZIER_SIZE; ++j) {
			curve->bernstein_weights_[i][j] = curve->binomial_coeff_[j] * t_powers[j] * one_minus_t_powers[BEZIER_SIZE - 1 - j];
			
			// 计算速度的Bernstein权重
			if(j < n){
				
				curve->bernstein_deriv_weights_[i][j] = curve->binomial_deriv_coeff_[j] * t_powers[j] * one_minus_t_powers[n - 1 - j];
				// if(i == 0 && j == 0){
				// 	printf("curve->binomial_deriv_coeff_:%f\n", curve->binomial_deriv_coeff_[j]);
				// 	printf("t_powers:%f\n", t_powers[j]);
				// 	printf("one_minus_t_powers:%f\n", one_minus_t_powers[n-1-j]);
				// 	printf("bernstein_deriv_weights_:%f\n", curve->bernstein_deriv_weights_[i][j]);
				// }
			}
		}
	}

    // calculate the fixed point based on robot state
    float x_start = footstep::init_state[0];
    float y_start = footstep::init_state[1];
    float vx_start = footstep::init_state[2];
    float vy_start = footstep::init_state[3];
    float theta_start = footstep::init_state[4];

    float x_goal = footstep::goal_state[0];
    float y_goal = footstep::goal_state[1];
    float vx_goal = footstep::goal_state[2];
    float vy_goal = footstep::goal_state[3];
    float theta_goal = footstep::goal_state[4];

	float sum_v = sqrtf(footstep::ux_ub * footstep::ux_ub + footstep::uy_ub + footstep::uy_ub);

    // Start point (P0)
    curve->control_points[0].x = x_start;
    curve->control_points[0].y = y_start;
	curve->control_points[0].z = theta_start;
	curve->is_point_xy_fixed[0] = 1;
	curve->is_theta_point_fixed[0] = 1;
    
    // // First control point (P1)
    // // 如果速度不为0，使用速度
    // if (abs(vx_start) > 1e-6 || abs(vy_start) > 1e-6) {
    //     // B'(0) = n(P1 - P0) = (vx_start, vy_start)
    //     // 所以 P1 = P0 + (vx_start/n, vy_start/n)
    //     curve->control_points[1].x = x_start + vx_start / static_cast<float>(n);
    //     curve->control_points[1].y = y_start + vy_start / static_cast<float>(n);
	// 	curve->is_point_xy_fixed[1] = 1;
    // } else {
    //     // 如果速度为0，使用角度
    //     // B'(0) = n(P1 - P0) = (cos(theta_0), sin(theta_0))
    //     // 所以 P1 = P0 + (cos(theta_0)/n, sin(theta_0)/n)
		
    //     curve->control_points[1].x = x_start + sum_v * __cosf(theta_start) / static_cast<float>(n);
    //     curve->control_points[1].y = y_start + sum_v * __sinf(theta_start) / static_cast<float>(n);
	// 	curve->is_point_xy_fixed[1] = 1;
    // }
    
    // // Second to last control point (Pn-1)
    // // 如果速度不为0，使用速度
    // if (abs(vx_goal) > 1e-6 || abs(vy_goal) > 1e-6) {
    //     // B'(1) = n(Pn - Pn-1) = (vx_end, vy_end)
    //     // 所以 Pn-1 = Pn - (vx_end/n, vy_end/n)
    //     curve->control_points[n-1].x = x_goal - vx_goal / static_cast<float>(n);
    //     curve->control_points[n-1].y = y_goal - vy_goal / static_cast<float>(n);
	// 	curve->is_point_xy_fixed[n-1] = 1;
    // } else {
    //     // 如果速度为0，使用角度
    //     // B'(1) = n(Pn - Pn-1) = (cos(theta_n), sin(theta_n))
    //     // 所以 Pn-1 = Pn - (cos(theta_n)/n, sin(theta_n)/n)
    //     // controlPoints.push_back({
    //     //     x_end - cos(theta_end)/n,
    //     //     y_end - sin(theta_end)/n
    //     // });
    //     curve->control_points[n-1].x = x_goal - sum_v * __cosf(theta_goal) / static_cast<float>(n);
    //     curve->control_points[n-1].y = y_goal - sum_v * __sinf(theta_goal) / static_cast<float>(n);
	// 	curve->is_point_xy_fixed[n-1] = 1;
    // }
    
    // End point (Pn)
    curve->control_points[n].x = x_goal;
    curve->control_points[n].y = y_goal;
	curve->control_points[n].z = theta_goal;
	curve->is_point_xy_fixed[n] = 1;
	curve->is_theta_point_fixed[n] = 1;
}

// __device__ __forceinline__ void GetTrajStateFromBezierBasedLookup(BezierCurve *curve, float *params, int t, int l, int r, int ll, int rr, float *state, bool convert){

//     float2 position{0.0f, 0.0f};
// 	float2 velocity{0.0f, 0.0f};
// 	const int n = BEZIER_SIZE - 1;

//     for (int i = 0; i < BEZIER_SIZE; ++i) {
//         float bernstein_t = curve->bernstein_weights_[t][i];
// 		position.x += bernstein_t * params[i + l];
// 		position.y += bernstein_t * params[i + ll];
		
// 		if(i < n){
// 			float deriv_ctrl_x = n * (params[i + 1 + l] - params[i + l]);
// 			float deriv_ctrl_y = n * (params[i + 1 + ll] - params[i + ll]);

// 			float bernstein_deriv_t = curve->bernstein_deriv_weights_[t][i];

// 			velocity.x += bernstein_deriv_t * deriv_ctrl_x;
//         	velocity.y += bernstein_deriv_t * deriv_ctrl_y;
// 		}
// 	}

//     if(!convert){
// 		// 存储状态
// 		int idx = t * 6;
// 		state[idx] = position.x;
// 		state[idx + 1] = position.y;
// 		state[idx + 2] = velocity.x;
// 		state[idx + 3] = velocity.y;
// 		state[idx + 4] = 0.0;		// theta
// 		state[idx + 5] = 0.0;		// radius for polar coordinate system
// 	}
//     else{
// 		PolarAllState2Cartesian(position, velocity, state, t);
// 	}
// }



BezierCurveManager::BezierCurveManager() { 
	hipMalloc(&curve_, sizeof(BezierCurve)); 
}
    
BezierCurveManager::~BezierCurveManager() { 
	hipFree(curve_); 
}
}
}
