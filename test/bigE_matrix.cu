#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define STATE_DIM 5  // E 是 5x5 矩阵

void construct_big_E(float* E, float* big_E, int N) {
    float *d_E, *d_Ek, *d_big_E;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // 分配 GPU 内存
    hipMalloc(&d_E, STATE_DIM * STATE_DIM * sizeof(float));
    hipMalloc(&d_Ek, STATE_DIM * STATE_DIM * sizeof(float));
    hipMalloc(&d_big_E, STATE_DIM * STATE_DIM * N * sizeof(float));

    // 复制 E 到 GPU
    hipMemcpy(d_E, E, STATE_DIM * STATE_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Ek, E, STATE_DIM * STATE_DIM * sizeof(float), hipMemcpyHostToDevice);

    // cuBLAS 计算矩阵幂次
    float alpha = 1.0f, beta = 0.0f;
    for (int k = 0; k < N; k++) {
        // 复制当前 Ek 到 big_E
        hipMemcpy(d_big_E + k * STATE_DIM * STATE_DIM, d_Ek, STATE_DIM * STATE_DIM * sizeof(float), hipMemcpyDeviceToDevice);

        // 计算 Ek+1 = E * Ek
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    STATE_DIM, STATE_DIM, STATE_DIM,
                    &alpha, d_E, STATE_DIM, d_Ek, STATE_DIM,
                    &beta, d_Ek, STATE_DIM);
    }

    // 复制结果回 CPU
    hipMemcpy(big_E, d_big_E, STATE_DIM * STATE_DIM * N * sizeof(float), hipMemcpyDeviceToHost);

    // 清理
    hipblasDestroy(handle);
    hipFree(d_E);
    hipFree(d_Ek);
    hipFree(d_big_E);
}

// 测试代码
int main() {
    int N = 30;  // 计算 100 步
    float E[STATE_DIM * STATE_DIM] = {
        1, 0, 0.513166, 0, 0,
        0, 1, 0, 0.513166, 0,
        0, 0, 1.89298, 0, 0,
        0, 0, 0, 1.89298, 0,
        0, 0, 0, 0, 1
    };

    float* big_E = new float[STATE_DIM * STATE_DIM * N];  // 存储大矩阵

    // 调用 cuBLAS 计算
    construct_big_E(E, big_E, N);

    // 打印部分结果
    for (int i = 0; i < N; i++) {  // 仅展示前 5 次幂
        std::cout << "E^" << (i+1) << ":\n";
        for (int j = 0; j < STATE_DIM; j++) {
            for (int k = 0; k < STATE_DIM; k++) {
                std::cout << big_E[i * STATE_DIM * STATE_DIM + j * STATE_DIM + k] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "-------------------" << std::endl;
    }

    delete[] big_E;
    return 0;
}