#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <memory>
#include <iomanip>
#include "cuda_bezier.cuh"

#define BEZIER_SIZE 7
#define NUM_STEPS 30

// 初始状态和目标状态
const float init_state[5] = {0.29357406, 0.29125562, -0.01193462, -0.01774755, 1.58432257};
const float goal_state[5] = {1.5, 2.8, 0, 0, 0};

using namespace cudaprocess;
using namespace bezier_curve;

#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

// 使用改进的逻辑生成控制点
void GenerateImprovedControlPoints(float* params, 
                                 float x_start, float y_start, float vx_start, float vy_start, float theta_start,
                                 float x_end, float y_end, float vx_end, float vy_end, float theta_end) {
    // 第一个控制点 - 起始位置
    params[0] = x_start;
    params[BEZIER_SIZE] = y_start;
    
    // 第二个控制点 - 受起始速度影响
    params[1] = x_start + vx_start / 6.0f;
    params[BEZIER_SIZE + 1] = y_start + vy_start / 6.0f;
    
    // 中间控制点 - 使用插值和角度影响
    for (int i = 2; i < 5; i++) {
        float t = i / 6.0f;
        // 基本位置通过线性插值
        float x = (1 - t) * x_start + t * x_end;
        float y = (1 - t) * y_start + t * y_end;
        
        // 添加角度的影响
        float theta_blend = (1 - t) * theta_start + t * theta_end;
        float radius = 0.3f * (1 - pow(2*t-1, 2)); // 抛物线影响因子
        
        x += radius * cos(theta_blend);
        y += radius * sin(theta_blend);
        
        params[i] = x;
        params[BEZIER_SIZE + i] = y;
    }
    
    // 倒数第二个控制点 - 受终止速度影响
    params[5] = x_end - vx_end / 6.0f;
    params[BEZIER_SIZE + 5] = y_end - vy_end / 6.0f;
    
    // 最后一个控制点 - 终止位置
    params[6] = x_end;
    params[BEZIER_SIZE + 6] = y_end;
}

// CUDA 核函数
__global__ void CUDA_BezierPosition(BezierCurve* d_curve, float* d_params, float* d_results) {
    int t_idx = threadIdx.x;
    
    // 使用查表法计算贝塞尔曲线上的点
    // float2 pos = GetBezierPositionBasedLookup(d_curve, d_params, t_idx, 0, BEZIER_SIZE-1, 
    //                                          BEZIER_SIZE, 2*BEZIER_SIZE-1, false);
    
    // // 存储结果
    // d_results[2 * t_idx] = pos.x;
    // d_results[2 * t_idx + 1] = pos.y;

    GetTrajStateFromBezierBasedLookup(d_curve, d_params, t_idx, 0, BEZIER_SIZE-1, 
                                                 BEZIER_SIZE, 2*BEZIER_SIZE-1, d_results);
}

int main() {
    int deviceCount;
    CHECK_CUDA_ERROR(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0) {
        fprintf(stderr, "No CUDA devices found\n");
        return -1;
    }
    
    // Print CUDA device info
    hipDeviceProp_t deviceProp;
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&deviceProp, 0));
    printf("Using CUDA device: %s\n", deviceProp.name);
    
    // 使用改进的方法生成控制点
    float params[2 * BEZIER_SIZE];
    GenerateImprovedControlPoints(params, 
                               init_state[0], init_state[1], init_state[2], init_state[3], init_state[4],
                               goal_state[0], goal_state[1], goal_state[2], goal_state[3], goal_state[4]);
    
    // 打印控制点
    printf("Control Points (Bezier curve of degree %d):\n", BEZIER_SIZE-1);
    for (int i = 0; i < BEZIER_SIZE; ++i) {
        printf("P%d: (%.6f, %.6f)\n", i, params[i], params[i + BEZIER_SIZE]);
    }
    
    // 分配 CUDA 设备内存
    float* d_params;
    float* d_results;

    CHECK_CUDA_ERROR(hipMalloc(&d_params, sizeof(float) * 2 * BEZIER_SIZE));
    CHECK_CUDA_ERROR(hipMalloc(&d_results, sizeof(float) * 5*(NUM_STEPS + 1)));

    std::shared_ptr<BezierCurveManager> bezier_curve_manager_ = std::make_shared<BezierCurveManager>();
    
    PrepareBinomial<<<1,1>>>(bezier_curve_manager_->curve_);
    CHECK_CUDA_ERROR(hipGetLastError());

    // 复制数据到设备
    CHECK_CUDA_ERROR(hipMemcpy(d_params, params, sizeof(float) * 2 * BEZIER_SIZE, hipMemcpyHostToDevice));

    // 启动 CUDA 计算
    CUDA_BezierPosition<<<1, NUM_STEPS + 1>>>(bezier_curve_manager_->curve_, d_params, d_results);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // 复制结果回主机
    std::vector<float> cuda_results(5 * (NUM_STEPS + 1));
    CHECK_CUDA_ERROR(hipMemcpy(cuda_results.data(), d_results, sizeof(float) * 5 * (NUM_STEPS + 1), hipMemcpyDeviceToHost));

    // 打印 CUDA 贝塞尔曲线上的点
    printf("\n%-5s %-15s %-15s %-15s %-15s %-15s\n", "t", "X", "Y", "Vx", "Vy", "Theta");
    printf("-----------------------------------------------------------------------\n");
    
    for (int i = 0; i <= NUM_STEPS; ++i) {
        float t = static_cast<float>(i) / NUM_STEPS;
        printf("%.3f %-15.6f %-15.6f %-15.6f %-15.6f %-15.6f\n", 
               t, 
               cuda_results[i * 5],     // x
               cuda_results[i * 5 + 1], // y
               cuda_results[i * 5 + 2], // vx
               cuda_results[i * 5 + 3], // vy
               cuda_results[i * 5 + 4]  // theta
        );
    }

    // 释放 CUDA 资源
    CHECK_CUDA_ERROR(hipFree(d_params));
    CHECK_CUDA_ERROR(hipFree(d_results));

    return 0;
}